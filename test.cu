#include "hip/hip_runtime.h"
 __global__ void calculate_forces(void *devX, void *devA)
 {   
    extern __shared__ float4[] shPosition;   
    float4 *globalX = (float4 *)devX;   
    float4 *globalA = (float4 *)devA;

    float4 myPosition;   
    
    int i, tile;   
    
    float3 acc = {0.0f, 0.0f, 0.0f};   
    
    int gtid = blockIdx.x * blockDim.x + threadIdx.x;   
    myPosition = globalX[gtid];   
    
    for (i = 0, tile = 0; i < N; i += p, tile++) 
    {     
        int idx = tile * blockDim.x + threadIdx.x;
        shPosition[threadIdx.x] = globalX[idx];
        
        __syncthreads();     
        
        acc = tile_calculation(myPosition, acc);     
        
        __syncthreads();       
    }   
    
    // Save the result in global memory for the integration step.
    float4 acc4 = {acc.x, acc.y, acc.z, 0.0f};   
    globalA[gtid] = acc4; 
} 